
#include "hip/hip_runtime.h"

#include <stdlib.h>
#include <stdio.h>

// The conventions for pointers is: if the value is stored in the device(GPU) it starts with d_, else if the value is stored
// in the host(CPU) the convention is to start with a h_

__global__ void kernel(float *d_out, float *d_in)
{
	
	// Create a function that returns pointers 
	int idx = threadIdx.x;
	float f = d_in[idx];
	// Output to a pointer from a float operation
	d_out[idx] = f * f * f;

}

int main()
{

	// Define the size of the array to utilize
	const int ARRAY_SIZE = 1000;
	// Calculate the size in bytes
	const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);
	
	// Create the input array on the host(CPU)
	float h_in[ARRAY_SIZE];
	for(int i = 0; i<ARRAY_SIZE; i++)
	{
		h_in[i] = float(i);
	}
	float h_out[ARRAY_SIZE];

	// Declare GPU memory pointers
	float *d_in;
	float *d_out;

	// Allocate GPU memory
	hipMalloc((void **)&d_in, ARRAY_BYTES);
	hipMalloc((void **)&d_out, ARRAY_BYTES);

	// Transfer the array to the GPU
	hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);

	// Launch the kernel, each block(left input) defines how many times the threads(right input) will be executed
	kernel <<<1, ARRAY_SIZE>>>(d_out, d_in);

	// Copy back the array to the host
	hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);

	// Print the resulting array
	for(int i = 0; i<ARRAY_SIZE; i++)
	{
	
		printf("%f", h_out[i]);
		printf(((i % 4) != 3) ? "\t" : "\n");

	}

	hipFree(d_in);
	hipFree(d_out);
	system("pause");
	return 0;

}
